#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <time.h> 
__global__ void my_strstr(char *str,char *sub_string,char ** position,int str_len,int sub_len,int num_sub)
{
  int id = threadIdx.x;
  //char *sub = &sub_string[id*sub_len];
  char *result = NULL; 
  char sub[24];
  //load sub in register,great improve
  for(int i=0;i<sub_len;++i){
    sub[i] = sub_string[id*sub_len+i];}
  //best case using Shared memory
  extern __shared__ char s_string[];
  //every thread has to fetch how many values from global memory to shared memory
  int each_num = str_len/blockDim.x;
  for(int i=0;i<each_num;++i){
    s_string[i*blockDim.x+id] = str[i*blockDim.x+id];}
  if( ((each_num*blockDim.x+id) < str_len) && (blockDim.x > each_num) )
    s_string[each_num*blockDim.x+id] = str[each_num*blockDim.x+id];
  __syncthreads();
  char *string = s_string;
//  char *length = s_string;
//  int i;
//  for(i = 0; length[i] != '\0'; i++);
//  printf("length is %d\n",i);
  char *a,*b;
  //b point to the sub address in register rather than in global memory
  b = sub;
  //result == NULL to judge if we find a match;rather than use goto or break in loop which harm the calculation
  int i;
  for(i = 0;(i < str_len)&&(result == NULL);i++){
    //printf("i am %d\n",id);
    a = string;
    while(*a++ == *b++){
      if(*(b+1) == '\0'){ 
        result = string;
      }
    }
    b = sub;
    ++string;
  }
  //coalesced global memory store, no effect since we only store once 
  position[id] = result;
//  printf("12213\n");
}



extern "C" int p_match( char* haystack, int LENGTH, long num_sub, int len, char* subs)
{
//  printf("Length is %d\n",LENGTH);
  int num_block,num_thread;
  if(num_sub < 512){
    num_block = 1;
    num_thread = num_sub;
  }
  else{
    num_block = num_sub / 512;
    num_thread = 512;
  }
  char *h_position[num_sub];

  char *d_string,*d_subs;
  char **d_position;
  hipMalloc((void**)&d_string,sizeof(char)*LENGTH);
  hipMalloc((void**)&d_subs,sizeof(char)*num_sub*len);
  hipMalloc((void***)&d_position,sizeof(char*)*num_sub);
  hipMemset(d_position,0,sizeof(char*)*num_sub);
  memset(h_position,0,sizeof(char*)*num_sub);
  const size_t smem = sizeof(char)*LENGTH;

  /*GPU*/
    hipMemcpy(d_string,haystack,sizeof(char)*LENGTH,hipMemcpyHostToDevice);
    hipMemcpy(d_subs,subs,sizeof(char)*num_sub*len,hipMemcpyHostToDevice);
//    printf("num_block is %d  and num_thread is %d\n",num_block,num_thread);
    my_strstr<<<num_block,num_thread,smem>>>(d_string,d_subs,d_position,LENGTH,len,num_sub);
    hipDeviceSynchronize();
    hipMemcpy(h_position,d_position,sizeof(char*)*num_sub,hipMemcpyDeviceToHost);

  return(0);
}
